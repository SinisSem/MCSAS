﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "../Types/BaseTypesRandom.h"

int main()
{
	MCSAS::Types::BaseTypes::StartRandom();
	for(int i = 0; i < 100; i++)
		float a = MCSAS::Types::BaseTypes::RandomBaseDispersion((float)10, (float)5);
    return 0;
}