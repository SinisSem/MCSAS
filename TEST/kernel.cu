﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "../Types/BaseTypes.h"

using namespace MCSAS;
using namespace Types;
using namespace BaseTypes;

int main()
{
	double A = Add(1,2);
	ComplexDoubleCartesian o1;
	o1.Re = 1;
	o1.Im = 2;
	ComplexDoubleCartesian o2;
	o2.Re = 3;
	o2.Im = 5;
	ComplexDoubleCartesian a = Add(o1,o2);
	a = Mul(o1,o2);


    return 0;
}