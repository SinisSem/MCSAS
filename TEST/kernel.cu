﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "../MatrixGenerator/MatrixGenerator.h"
#include "../Types/COO_matrix.h"
#include <vector>
#include "../Types/IO/MatrixMarket.h"
//#include "../Problem/Problem.h"
#include <iostream>
#include "../Types/CUDATypes/Array2D.h"
#include "../Problem/Problem.h"

using namespace MCSAS;

int main()
{
//	MCSAS::MatrixGenerator::MGTask<double> Task = MCSAS::MatrixGenerator::GetStandardTask<double>(1);
//	MCSAS::MatrixGenerator::CMatrixGenerator<double> MG;
//	MG.GenerateInternal(Task);
	try{
		MCSAS::Types::COO_matrix<float> A;// = MG.GetCOOMatrix();
		std::vector<float> B;// = MG.GetRightVector();

		MCSAS::Types::IO::ReadMMFile(A, "mat0.txt");
		MCSAS::Types::IO::ReadMMFile(B, "vec0.txt");

		//MCSAS::MatrixGenerator::CMatrixGenerator<double> MG;

		MCSAS::Problem::CProblemReal<float> Prb;
		Prb.FormulateProblem(A, B);

//		std::vector<MCSAS::Types::CUDATypes::CArray2D<double>> AA;

//		MCSAS::Problem::CProblem<double> Problem;
	
//		Problem.FormulateProblem(A, B);
	}
	catch(MCSAS::MCSASException Ex)
	{
		std::cout<<Ex.what(); 
	}

    return 0;
}