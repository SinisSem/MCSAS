﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "../MatrixGenerator/MatrixGenerator.h"
#include "../Types/COO_matrix.h"
#include <vector>
#include "../Types/IO/MatrixMarket.h"
#include "../Problem/Problem.h"
#include <iostream>

using namespace MCSAS;

int main()
{
//	MCSAS::MatrixGenerator::MGTask<double> Task = MCSAS::MatrixGenerator::GetStandardTask<double>(1);
//	MCSAS::MatrixGenerator::CMatrixGenerator<double> MG;
//	MG.GenerateInternal(Task);
	try{
		MCSAS::Types::COO_matrix<double> A;// = MG.GetCOOMatrix();
		std::vector<double> B;// = MG.GetRightVector();

		MCSAS::Types::IO::ReadMMFile(A, "mat0.txt");
		MCSAS::Types::IO::ReadMMFile(B, "vec0.txt");

		MCSAS::Problem::CProblem<double> Problem;
	
		Problem.FormulateProblem(A, B);
	}
	catch(MCSAS::MCSASException Ex)
	{
		std::cout<<Ex.what(); 
	}

    return 0;
}