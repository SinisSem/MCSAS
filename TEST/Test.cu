﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "../MatrixGenerator/MatrixGenerator.h"
#include "../Types/COO_matrix.h"
#include <vector>
#include "../Types/IO/MatrixMarket.h"

using namespace MCSAS;

int main()
{
	MCSAS::MatrixGenerator::MGTask<double> Task = MCSAS::MatrixGenerator::GetStandardTask<double>(1);
	MCSAS::MatrixGenerator::CMatrixGenerator<double> MG;
	MG.SetTask(Task);
	MCSAS::Types::COO_matrix<double> A = MG.GetCOOMatrix();
	std::vector<double> B = MG.GetRightVector();

	MCSAS::Types::IO::WriteMMFile(A, "mat1.txt");
	MCSAS::Types::IO::WriteMMFile(B, "vec1.txt");

	MCSAS::Types::IO::ReadMMFile(A, "mat0.txt");
	MCSAS::Types::IO::ReadMMFile(B, "vec0.txt");

    return 0;
}